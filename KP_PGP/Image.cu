#include "hip/hip_runtime.h"
#include "Image.cuh.cu"

#include <fstream>

#include "utils.cuh.cu"
#include "./dummy_helper.cuh.cu"

namespace RayTracing
{

Image::Image(const std::string &fileName)
{
    std::ifstream file(fileName);
    
    file.read(reinterpret_cast<char*>(&m_width), sizeof(m_width));
    file.read(reinterpret_cast<char*>(&m_height), sizeof(m_height));

    for (int i = 0; i < m_width * m_height; ++i)
    {
        unsigned char r = 0,
                      g = 0,
                      b = 0,
                      a = 0;

        file.read(reinterpret_cast<char*>(&r), sizeof(r));
        file.read(reinterpret_cast<char*>(&g), sizeof(g));
        file.read(reinterpret_cast<char*>(&b), sizeof(b));
        file.read(reinterpret_cast<char*>(&a), sizeof(a));

        buffer.push_back(Color{
            r / 255.0f,
            g / 255.0f,
            b / 255.0f
        });
    }
}

template<>
void Image::Init<true>()
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    checkCudaErrors(hipMallocArray(
        &m_buffer_d, 
        &channelDesc, 
        m_width,
        m_height
    ));

    checkCudaErrors(hipMemcpyToArray(
        m_buffer_d,
        0,
        0,
        buffer.data(),
        buffer.size() * sizeof(float4),
        hipMemcpyHostToDevice
    ));
    
    memset(&m_cudaTextureResourceDesc, 0, sizeof(m_cudaTextureResourceDesc));

    m_cudaTextureResourceDesc.resType            = hipResourceTypeArray;
    m_cudaTextureResourceDesc.res.array.array    = m_buffer_d;

    memset(&m_cudaTextureDesc, 0, sizeof(m_cudaTextureDesc));

    m_cudaTextureDesc.normalizedCoords = true;
    m_cudaTextureDesc.filterMode       = hipFilterModePoint;
    m_cudaTextureDesc.addressMode[0] = hipAddressModeClamp;
    m_cudaTextureDesc.addressMode[1] = hipAddressModeClamp;
    m_cudaTextureDesc.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(
        &cudaTexture,
        &m_cudaTextureResourceDesc,
        &m_cudaTextureDesc,
        NULL
    ));
}

template<>
hipTextureObject_t Image::GetResource<true>()
{
    return cudaTexture;
}

template<>
Image Image::GetResource<false>()
{
    return *this;
}

template<>
void Image::Init<false>() {}

Color Image::GetColor(const float u, const float v) const
{
    int w = Clamp(u * (m_width - 1), 0, (m_width - 1)),
        h = Clamp(v * (m_height - 1), 0, (m_height - 1));

    return buffer[w + m_width * h];
}

void Image::Deinit()
{
    if (m_buffer_d != nullptr)
    {
        checkCudaErrors(hipFreeArray(m_buffer_d));
        checkCudaErrors(hipDestroyTextureObject(cudaTexture));
    }
}


} // RayTracing
