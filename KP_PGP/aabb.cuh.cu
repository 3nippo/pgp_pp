#include "hip/hip_runtime.h"
#pragma once

#include <algorithm>
#include <cmath>

#include "Vector3.cuh.cu"
#include "Ray.cuh.cu"

namespace RayTracing
{

class aabb 
{
private:
    Point3 m_min, m_max;
public:
    aabb() {}
    aabb(const Point3& min, const Point3& max) 
        : m_min(min), m_max(max)
    {}

    Point3 min() const { return m_min; }
    Point3 max() const { return m_max; }
    
    __host__ __device__
    bool Hit(const Ray& r, double t_min, double t_max) const 
    {
        for (int a = 0; a < 3; a++) 
        {
            auto t0 = fminf((m_min[a] - r.origin[a]) / r.direction[a],
                           (m_max[a] - r.origin[a]) / r.direction[a]);
            auto t1 = fmaxf((m_min[a] - r.origin[a]) / r.direction[a],
                           (m_max[a] - r.origin[a]) / r.direction[a]);
            t_min = fmaxf(t0, t_min);
            t_max = fminf(t1, t_max);
            if (t_max <= t_min)
                return false;
        }
        return true;
    }

    static aabb SurroundingBox(const aabb &a, const aabb &b)
    {
        return aabb{
            Point3{
                std::min(a.m_min.d.x, b.m_min.d.x),
                std::min(a.m_min.d.y, b.m_min.d.y),
                std::min(a.m_min.d.z, b.m_min.d.z),
            },
            Point3{
                std::max(a.m_max.d.x, b.m_max.d.x),
                std::max(a.m_max.d.y, b.m_max.d.y),
                std::max(a.m_max.d.z, b.m_max.d.z),
            }
        };
    }
};

} // RayTracing
