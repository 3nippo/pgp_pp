#include "hip/hip_runtime.h"
#pragma once

#include "TriangleFace.cuh.cu"
#include "utils.cuh.cu"
#include "dummy_helper.cuh.cu"
#include "PolygonsManager.cuh.cu"


#include <vector>
#include <limits>

namespace RayTracing
{

struct BVHNode
{
aabb box;

size_t left;
size_t right;
size_t polygonIndex;

static void FromFaces(
    std::vector<MappedTriangleFace>& faces,
    std::vector<BVHNode>& nodes,
    size_t start,
    size_t end
) 
{
    nodes.emplace_back(BVHNode(faces, nodes, start, end));   
}

BVHNode(
    std::vector<MappedTriangleFace>& faces,
    std::vector<BVHNode>& nodes,
    size_t start,
    size_t end
)
{
    int axis = GenRandom(0, 2.999);

    int spanCount = end - start;
    
    polygonIndex = std::numeric_limits<size_t>::max();

    auto cmp = [axis](const MappedTriangleFace& a, const MappedTriangleFace& b)
    {
        return aabb::Compare(a.BoundingBox(), b.BoundingBox(), axis);
    };

    if (spanCount == 1)
    {
        polygonIndex = start;
        box = faces[polygonIndex].BoundingBox();

        return;
    }

    if (spanCount == 2)
    {
        nodes.emplace_back();
        nodes.emplace_back();

        if (cmp(faces[start], faces[start+1]))
        {
            left = nodes.size() - 2;
            nodes[left].polygonIndex = start;

            right = nodes.size() - 1;
            nodes[right].polygonIndex = start+1;
        }
        else
        {
            right = nodes.size() - 2;
            nodes[right].polygonIndex = start;

            left = nodes.size() - 1;
            nodes[left].polygonIndex = start+1;
        }
    }
    else
    {
        std::sort(faces.begin() + start, faces.begin() + end, cmp);

        size_t mid = start + spanCount/2;
        
        nodes.emplace_back(faces, nodes, start, mid);
        left = nodes.size() - 1;

        nodes.emplace_back(faces, nodes, mid, end);
        right = nodes.size() - 1;
    }

    box = aabb::SurroundingBox(
        nodes[left].box,
        nodes[right].box
    );
}
};

template<bool isGPU>
class BVH
{

};

template<>
class BVH<false>
{
protected:
    PolygonsManager<false> m_polygonsManager;
    std::vector<BVHNode> m_nodes;
public:
    BVH(
        std::vector<MappedTriangleFace>& faces
    )
    {
        BVHNode::FromFaces(faces, m_nodes, 0, faces.size());
    }

    void InitBeforeRender() {}
    void DeinitAfterRender() {}
    
    bool Hit(
        const Ray &ray, 
        const float tMin,
        HitRecord &hitRecord
    ) const
    {
        return HitHelper(ray, tMin, hitRecord, m_nodes.size() - 1);
    }

protected:
    bool HitHelper(
        const Ray &ray, 
        const float tMin,
        HitRecord &hitRecord,
        size_t index
    ) const
    {
        if (!m_nodes[index].box.Hit(ray, tMin, hitRecord.t))
            return false;

        if (m_nodes[index].polygonIndex != std::numeric_limits<size_t>::max())
        {
            return m_polygonsManager.Hit(
                ray,
                tMin,
                hitRecord,
                index
            );
        }

        return HitHelper(ray, tMin, hitRecord, m_nodes[index].left)
            || HitHelper(ray, tMin, hitRecord, m_nodes[index].right);
    }
};

template<>
class BVH<true> : public BVH<false>
{
private:
    PolygonsManager<true> m_polygonsManager;
    CudaMemoryLogic<BVHNode> m_nodes_d;

public:
    using BVH<false>::BVH;

    
    void InitBeforeRender() 
    {
        m_nodes_d.memcpy(this->m_nodes.data(), hipMemcpyHostToDevice);
        this->m_nodes.clear();
    }
    void DeinitAfterRender() 
    {
        m_nodes_d.dealloc();
    }
    
    __device__
    bool Hit(
        const Ray &ray, 
        const float tMin,
        HitRecord &hitRecord
    ) const
    {
        return HitHelper(ray, tMin, hitRecord, m_nodes_d.count - 1);
    }
private:
    __device__
    bool HitHelper(
        const Ray &ray, 
        const float tMin,
        HitRecord &hitRecord,
        size_t index
    ) const
    {
        if (!m_nodes_d.get()[index].box.Hit(ray, tMin, hitRecord.t))
            return false;

        if (m_nodes_d.get()[index].polygonIndex != std::numeric_limits<size_t>::max())
        {
            return m_polygonsManager.Hit(
                ray,
                tMin,
                hitRecord,
                index
            );
        }

        return HitHelper(ray, tMin, hitRecord, m_nodes_d.get()[index].left)
            || HitHelper(ray, tMin, hitRecord, m_nodes_d.get()[index].right);
    }
};

} // namespace RayTracing
