#pragma once

#include "Material.cuh.cu"
#include "HitRecord.cuh.cu"
#include "Ray.cuh.cu"
#include "utils.cuh.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace RayTracing
{

class Lambertian : public Material
{
protected:
    const Texture* const * const m_albedo;
    hiprandState *m_states;
    
public:
    __host__ __device__
    Lambertian(
        const float transparency,
        const float reflectance,
        const Texture* const * const albedo,
        hiprandState *states
    )
        : Material(transparency, reflectance),
          m_albedo(albedo),
          m_states(states)
    {}

private:
    __host__ __device__
    Vector3 RandomUnitSphereSurfaceVector() const
    {
        #ifdef __CUDA_ARCH__
        
        int id = threadIdx.x + blockDim.x * blockIdx.x;

        while (true)
        {
            Vector3 v{
                hiprand_uniform(m_states + id) * 2 - 1,
                hiprand_uniform(m_states + id) * 2 - 1,
                hiprand_uniform(m_states + id) * 2 - 1
            };

            if (v.LengthSquared() > 1)
                continue;

            return v.UnitVector();
        }

        #else

        while (true)
        {
            Vector3 v{
                GenRandom(-1, 1),
                GenRandom(-1, 1),
                GenRandom(-1, 1),
            };

            if (v.LengthSquared() > 1)
                continue;

            return v.UnitVector();
        }

        #endif
    }
    
    __host__ __device__
    virtual bool Scatter(
        const Ray &ray,
        const HitRecord &hitRecord,
        Color &attenuation,
        Ray &scattered
    ) const override
    {
        Vector3 scatterDir = hitRecord.normal + RandomUnitSphereSurfaceVector();

        if (scatterDir.NearZero())
            scatterDir = hitRecord.normal;

        scattered = Ray(
            hitRecord.point,
            scatterDir
        );

        attenuation = (*m_albedo)->GetColor(hitRecord.u, hitRecord.v);

        return true;
    }
};

} // namespace RayTracing
