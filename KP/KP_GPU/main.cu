#include "hip/hip_runtime.h"
#include <string>
#include <cmath>
#include <iostream>
#include <cstdlib>

#include "RayTracer.cuh.cu"
#include "PolygonsManager.cuh.cu"
#include "FigureConstructor.cuh.cu"
#include "Camera.cuh.cu"
#include "Vector3.cuh.cu"
#include "Lambertian.cuh.cu"
#include "Metallic.cuh.cu"
#include "Image.cuh.cu"
#include "ImageTexture.cuh.cu"
#include "Texture.cuh.cu"
#include "DiffuseLight.cuh.cu"
#include "DummyAllocs.cuh.cu"
#include "Config.cuh.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace
{

__global__ 
void InitStates(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(42, id, 0, state + id);
}

int PrintDefaultConfig()
{
    int err = system("cat ./default_config");
    
    if (err)
    {
        std::cout << "You probably lost config >:(" << std::endl;
    }

    return err;
}

CudaMemoryLogic<hiprandState> states;

template<
    template<typename...> class ObjectAllocator,
    bool isGPU,
    typename TextureResource
>
void Logic(
    const RayTracing::Config &config
)
{
    using namespace RayTracing;

    PolygonsManager<isGPU> polygonsManager;
    
    if (isGPU)
    {
        states.alloc(GRID_SIZE * BLOCK_SIZE);

        InitStates<<<GRID_SIZE, BLOCK_SIZE>>>(states.get());
    }

    // Figure A

    ObjectAllocator<SolidTexture, Texture, Color> pinkTexture(Color(1, 0.07, 0.57));

    ObjectAllocator<Lambertian, Material, float, float, Texture**, hiprandState*> pinkMaterial(
        0,
        1,
        pinkTexture.ptr,
        states.get()
    );

    ObjectAllocator<SolidTexture, Texture, Color> mirrorTexture(config.A.color); 
    
    ObjectAllocator<Metallic, Material, float, float, Texture**> mirrorMaterial(
        config.A.transparency,
        config.A.reflectance,
        mirrorTexture.ptr
    );
    
    ObjectAllocator<SolidTexture, Texture, Color> edgeLightTexture(Color(2, 2, 2));

    ObjectAllocator<DiffuseLight, Material, Texture**> edgeLightMaterial(edgeLightTexture.ptr);
    
    FigureConstructor<FigureId::FancyCube, isGPU>::ConstructFigure(
        polygonsManager,
        {
            mirrorMaterial.ptr,
            pinkMaterial.ptr,
            edgeLightMaterial.ptr
        },
        config.A.origin,
        config.A.radius,
        config.A.edgeLightsNum
    );

    // LightSources

    std::vector<ObjectAllocator<
        RayTracing::SolidTexture, 
        RayTracing::Texture, 
        RayTracing::Color
    >> lightSourcesTextures;

    lightSourcesTextures.reserve(config.lightSourcesNum * 10);

    std::vector<ObjectAllocator<
        RayTracing::DiffuseLight, 
        RayTracing::Material, 
        RayTracing::Texture**
    >> lightSourcesMaterials;

    lightSourcesMaterials.reserve(config.lightSourcesNum * 10);

    for (int i = 0; i < config.lightSourcesNum; ++i)
    {
        lightSourcesTextures.emplace_back(
            config.lightSources[i].color
        );

        lightSourcesMaterials.emplace_back(
            lightSourcesTextures.back().ptr
        );

        FigureConstructor<FigureId::LightSource, isGPU>::ConstructFigure(
            polygonsManager,
            { lightSourcesMaterials[i].ptr },
            config.lightSources[i].origin,
            config.lightSources[i].radius,
            0
        );
    }

    // Floor

    Image floorImage(config.floorData.texturePath);
    
    floorImage.Init<isGPU>();

    ObjectAllocator<ImageTexture<isGPU>, Texture, TextureResource, Color> floorTexture(
        floorImage.GetResource<isGPU, TextureResource>(),
        config.floorData.color
    );

    ObjectAllocator<Lambertian, Material, float, float, Texture**, hiprandState*> floorMaterial(
        0, 
        config.floorData.reflectance,
        floorTexture.ptr,
        states.get()
    );

    FigureConstructor<FigureId::Floor, isGPU>::ConstructFigureByPoints(
        polygonsManager,
        { floorMaterial.ptr },
        config.floorData.A,
        config.floorData.B,
        config.floorData.C,
        config.floorData.D
    );

    polygonsManager.CompleteAdding();
    
    RayTracer rayTracer(config, 0, config.framesNum);
    
    rayTracer.RenderFrames(polygonsManager);
    
    floorImage.Deinit();
    polygonsManager.Deinit();

    if (isGPU)
    {
        states.dealloc();
    }
}

} // namespace

int main(int argc, char **argv)
{
try
{
    if (argc == 2 && std::string(argv[1]) == "--default")
    {
        return PrintDefaultConfig();
    }

    bool useGPU = true;

    if (argc == 2 && std::string(argv[1]) == "--cpu")
        useGPU = false;
    
    RayTracing::Config config;

    std::cin >> config;

    if (useGPU)
        Logic<RayTracing::CudaHeapObject, true, hipTextureObject_t>(config);
    else
        Logic<RayTracing::HeapObject, false, RayTracing::Image>(config);
}
catch (std::runtime_error &err)
{
    std::cout << err.what() << std::endl;
}

    return 0;
}
