#pragma once

#include <vector>
#include <string>

#include "Vector3.cuh.cu"

namespace RayTracing
{
class Image
{
private:
    int m_width;
    int m_height;

    hipResourceDesc m_cudaTextureResourceDesc;
    hipTextureDesc m_cudaTextureDesc;
    hipArray *m_buffer_d;

public:
    std::vector<Color> buffer;
    hipTextureObject_t cudaTexture;

public:
    Image(const std::string &fileName);

    ~Image();
};
};
